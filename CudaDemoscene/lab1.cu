#include "lab1.h"

// Cuda ���� .h
#include <hip/hip_runtime.h>
#include <>

#include "colorTransfer.cuh"

#include "SceneSetting.h"
#include "SceneSetting.cuh"

#include "RainClass.h"
#include "Rain_kernels.cuh"

#include "Ground.h"
#include "Ground_kernels.cuh"

#include "DigtalAlphabet_kernels.cuh"
#include "DevelopWord.h"

#include "Building.h"
#include "Building_kernels.cuh"

#pragma region ��l�Ƴ]�w
static const int NFRAME = 360;										// �`�@�� Frame ��

static const int ThreadSize = 12;									// Thread �� Size

struct Lab1VideoGenerator::Impl {
	int t = 0;
};
Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
}
Lab1VideoGenerator::~Lab1VideoGenerator() {}
void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) 
{
	info.w = SceneSetting::Width;
	info.h = SceneSetting::Height;
	info.n_frame = NFRAME;

	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};
#pragma endregion

void Lab1VideoGenerator::Generate(uint8_t *yuv) 
{
	#pragma region ��l��
	hipMemset(yuv, 0, SceneSetting::Width * SceneSetting::Height);
	hipMemset(yuv + SceneSetting::Width * SceneSetting::Height, 128, SceneSetting::Width * SceneSetting::Height / 2);
	Building::Init();
	#pragma endregion
	#pragma region Camera �첾
	if (6 * 24 <= impl->t && impl->t < 7 * 24)
		Camera::offsetY -= Camera::MoveSpeed;
	#pragma endregion
	#pragma region RainClass ������
	RainClass::GenerateRain();
	RainClass::MoveRain();
	RainClass::MoveRainDrop();
	RainClass::AddTime();
	Building::AddTime();
	
	#pragma region �P�_�B�O�_������F��
	Building::CheckIsHit();
	Ground::CheckIsHit();
	#pragma endregion		
	#pragma region Rain �� Cuda Part
	RainInfo *HostRainData, *DeviceRainData;
	HostRainData = RainClass::rainArray.data();
	size_t RainInfoSize = sizeof(RainInfo) * RainClass::rainArray.size();

	// Copy �W Cuda
	hipMalloc(&DeviceRainData, RainInfoSize);
	hipMemcpy(DeviceRainData, HostRainData, RainInfoSize, hipMemcpyHostToDevice);
	#pragma endregion
	#pragma region RainDrop �� Cuda Part
	RainDrop *HostRainDropData, *DeviceRainDropData;
	HostRainDropData = RainClass::rainDropArray.data();
	size_t RainDropInfoSize = sizeof(RainDrop) * RainClass::rainDropArray.size();

	// Copy �W Cuda
	hipMalloc(&DeviceRainDropData, RainDropInfoSize);
	hipMemcpy(DeviceRainDropData, HostRainDropData, RainDropInfoSize, hipMemcpyHostToDevice);
	#pragma endregion
	#pragma endregion
	#pragma region �r������
	char *HostDevelopWord, *DeviceDevelopWord;
	HostDevelopWord = &DevelopWord::word[0u];
	size_t DevelopWordSize = sizeof(char) * DevelopWord::word.size();

	// Copy �W Cuda
	hipMalloc(&DeviceDevelopWord, DevelopWordSize);
	hipMemcpy(DeviceDevelopWord, HostDevelopWord, DevelopWordSize, hipMemcpyHostToDevice);

	char *HostStartTime, *DeviceStartTime;
	string currentTime = DevelopWord::FormatStartTime();
	HostStartTime = &currentTime[0u];
	size_t StartTimeSize = sizeof(char) * currentTime.size();

	// Copy �W Cuda
	hipMalloc(&DeviceStartTime, StartTimeSize);
	hipMemcpy(DeviceStartTime, HostStartTime, StartTimeSize, hipMemcpyHostToDevice);
	#pragma endregion
	#pragma region �Фl����
	Line *HostLineData, *DeviceLineData;
	HostLineData = Building::lineSet.data();
	size_t LineSize = sizeof(Line) * Building::lineSet.size();

	// Copy �W Cuda
	hipMalloc(&DeviceLineData, LineSize);
	hipMemcpy(DeviceLineData, HostLineData, LineSize, hipMemcpyHostToDevice);

	// ��r
	char *HostBuildingText, *DeviceBuildingText;
	HostBuildingText = &Building::text[0u];
	size_t BuildingTextSize = sizeof(char) * Building::text.size();

	// Copy �W Cuda
	hipMalloc(&DeviceBuildingText, BuildingTextSize);
	hipMemcpy(DeviceBuildingText, HostBuildingText, BuildingTextSize, hipMemcpyHostToDevice);

	// �}��r
	char *HostBrokenText, *DeviceBrokenText;
	HostBrokenText = &Building::BrokenText[0u];
	size_t BrokenTextSize = sizeof(char) * Building::BrokenText.size();

	// Copy �W Cuda
	hipMalloc(&DeviceBrokenText, BrokenTextSize);
	hipMemcpy(DeviceBrokenText, HostBrokenText, BrokenTextSize, hipMemcpyHostToDevice);
	#pragma endregion
	#pragma region �e�F��
	#pragma region �B
	int RainBlockSize = RainClass::rainArray.size() / ThreadSize + 1;
	RenderRain << <RainBlockSize, ThreadSize >> > (yuv, DeviceRainData, RainClass::rainArray.size(), Camera::offsetY, RainClass::Brightness());
	
	int RainDropBlockSize = RainClass::rainDropArray.size() / ThreadSize + 1;
	RenderRainDrop << <RainDropBlockSize, ThreadSize >> > (yuv, DeviceRainDropData, RainClass::rainDropArray.size(), Camera::offsetY, RainClass::Brightness());
	#pragma endregion
	#pragma region �a��
	int GroundBlockSize = SceneSetting::Width / ThreadSize + 1;
	RenderGound << <GroundBlockSize, ThreadSize >> > (yuv, Camera::offsetY);
	#pragma endregion
	#pragma region �r��
	int AlphabetBlockSize = DevelopWord::word.size() / ThreadSize + 1;
	RenderAlphabet << <AlphabetBlockSize, ThreadSize >> > (yuv, DevelopWord::GetPos(), DeviceDevelopWord, DevelopWord::word.size(), DevelopWord::Brightness(), Camera::offsetY);

	int StartTimeBlockSize = currentTime.size() / ThreadSize + 1;
	RenderAlphabet << <StartTimeBlockSize, ThreadSize >> > (yuv, DevelopWord::GetStartTimePos(), DeviceStartTime, currentTime.size(), DevelopWord::FormatStartTimeBrightness(), Camera::offsetY);
	#pragma endregion
	#pragma region �Фl
	int BuildingBlockSize = Building::lineSet.size() / ThreadSize + 1;
	RenderLine << <BuildingBlockSize, ThreadSize >> > (yuv, DeviceLineData, Building::lineSet.size(), Building::BuildingBrightness(), Camera::offsetY);

	int BuildingTextBlockSize = Building::text.size() / ThreadSize + 1;
	RenderAlphabet << <AlphabetBlockSize, ThreadSize >> > (yuv, Building::GetTextPos(), DeviceBuildingText, Building::text.size(), 1, Camera::offsetY);

	int BrokenTextBlockSize = Building::BrokenText.size() / ThreadSize + 1;
	RenderAlphabet << <BrokenTextBlockSize, ThreadSize >> > (yuv, Building::GetBrokenTextPos(), DeviceBrokenText, Building::BrokenText.size(), Building::BrokenTextBrightness(), Camera::offsetY);
	#pragma endregion
	#pragma endregion
	#pragma region �R���Ծ��O����
	#pragma region �B
	hipFree(DeviceRainData);
	hipFree(DeviceRainDropData);
	#pragma endregion
	#pragma region �r
	hipFree(DeviceDevelopWord);
	#pragma endregion
	#pragma region �Фl
	hipFree(DeviceLineData);
	hipFree(DeviceBuildingText);
	hipFree(DeviceBrokenText);
	#pragma endregion
	#pragma endregion

	// �ɶ���s
	++(impl->t);
}
